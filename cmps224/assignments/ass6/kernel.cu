#include "hip/hip_runtime.h"

#include "common.h"

#include "timer.h"

#define BLOCK_DIM 1024

__global__ void reduce_kernel(float *input, float *sum, unsigned int N) {}

float reduce_gpu(float *input, unsigned int N) {

  Timer timer;

  // Allocate memory
  startTime(&timer);
  float *input_d;
  hipMalloc((void **)&input_d, N * sizeof(float));
  float *sum_d;
  hipMalloc((void **)&sum_d, sizeof(float));
  hipDeviceSynchronize();
  stopTime(&timer);
  printElapsedTime(timer, "Allocation time");

  // Copy data to GPU
  startTime(&timer);
  hipMemcpy(input_d, input, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemset(sum_d, 0, sizeof(float));
  hipDeviceSynchronize();
  stopTime(&timer);
  printElapsedTime(timer, "Copy to GPU time");

  // Call kernel
  startTime(&timer);
  const unsigned int numThreadsPerBlock = BLOCK_DIM;
  const unsigned int numElementsPerBlock = 2 * numThreadsPerBlock;
  const unsigned int numBlocks =
      (N + numElementsPerBlock - 1) / numElementsPerBlock;
  reduce_kernel<<<numBlocks, numThreadsPerBlock>>>(input_d, sum_d, N);
  hipDeviceSynchronize();
  stopTime(&timer);
  printElapsedTime(timer, "Kernel time", GREEN);

  // Copy data from GPU
  startTime(&timer);
  float sum;
  hipMemcpy(&sum, sum_d, sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  stopTime(&timer);
  printElapsedTime(timer, "Copy from GPU time");

  // Free memory
  startTime(&timer);
  hipFree(input_d);
  hipFree(sum_d);
  hipDeviceSynchronize();
  stopTime(&timer);
  printElapsedTime(timer, "Deallocation time");

  return sum;
}
